#include "hip/hip_runtime.h"
#ifndef __SPH_KERNEL_CU__
#define __SPH_KERNEL_CU__

__constant__ SPH::FluidParams    cudaFluidParams;
__constant__ SPH::PrecalcParams  cudaPrecalcParams;

#include "boundary_walls.cu"
#include "grid.cuh"
#include "grid_utils.cu"
#include "sph_density.cu"
#include "sph_force.cu"
#include "sph_neighbours.cu"
#include "colors.cu"

namespace SPH {

    namespace Kernel {

        using namespace Grid::Utils;

        ////////////////////////////////////////////////////////////////////////

        template<class D>
        __global__ void integrate(
            int numParticles,
            float deltaTime,
            float3 gravity,
            D data,
            D sortedData,
            GridData gridData
        ) {
            int index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
            if (index >= numParticles) {
                return;
            }

            float3 position = make_float3(sortedData.position[index]);
            float3 velocity = make_float3(sortedData.velocity[index]);
            float3 veleval = make_float3(sortedData.veleval[index]);

            float3 force = make_float3(sortedData.force[index]);
            float pressure = sortedData.pressure[index];

            float3 externalForce = gravity;

            // add no-penetration force due to "walls"
            externalForce += Boundary::Walls::calculateWallsNoPenetrationForce(
                    position, veleval,
                    cudaGridParams.min,
                    cudaGridParams.max,
                    cudaFluidParams.boundaryDistance,
                    cudaFluidParams.boundaryStiffness,
                    cudaFluidParams.boundaryDampening);

            // add no-slip force due to "walls"
            /*externalForce += Boundary::Walls::calculateWallsNoSlipForce(
                    position, veleval, force + externalForce,
                    cudaGridParams.min,
                    cudaGridParams.max,
                    cudaFluidParams.boundaryDistance,
                    cudaFluidParams.frictionKinetic/deltaTime,
                    cudaFluidParams.frictionStaticLimit);
            */
            force += externalForce;

            float speed = length(force);

            if (speed > cudaFluidParams.velocityLimit) {
                force *= cudaFluidParams.velocityLimit / speed;
            }

            float3 vnext = velocity + force * deltaTime;
            veleval = (velocity + vnext) * 0.5f;
            velocity = veleval;

            position += vnext * deltaTime;

            uint sortedIndex = gridData.index[index];

            /*if ((position.y - EPSILON) <= cudaGridParams.min.y) {
                position.y =  cudaGridParams.min.y + EPSILON;
            }

            if ((position.y + EPSILON) >= cudaGridParams.max.y) {
                position.y =  cudaGridParams.max.y - EPSILON;
            }*/


            data.position[sortedIndex] = make_float4(position, 1.0f);
            data.velocity[sortedIndex] = make_float4(velocity, 1.0f);
            data.veleval[sortedIndex] = make_float4(veleval, 1.0f);

            if (cudaFluidParams.dynamicColoring) {
                //float3 color = (position - cudaGridParams.min) / cudaGridParams.size;
                float3 color =
                    Colors::calculateColor(
                        Colors::HSVBlueToRed,
                        Colors::Velocity,
                        vnext,
                        pressure,
                        force
                    );
                data.color[sortedIndex] = make_float4(color, 1.0f);
            }

        }

        ////////////////////////////////////////////////////////////////////////


        // TODO this is the same for classical simulator, so place somewhere
        // where general codes are
        template<class D>
        __global__ void update (
            uint numParticles,
            D unsortedData,
            D sortedData,
            GridData gridData
        ) {
            uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
            if (index >= numParticles) {
                return;
            }

            extern __shared__ uint sharedHash[]; // blockSize + 1 elements

            uint hash = gridData.hash[index];

            sharedHash[threadIdx.x+1] = hash;

            if (index > 0 && threadIdx.x  == 0) {
                sharedHash[0] = gridData.hash[index-1];
            }

            __syncthreads();

            if (index == 0 || hash != sharedHash[threadIdx.x]) {
                gridData.cellStart[hash] = index;

                if (index > 0) {
                    gridData.cellStop[sharedHash[threadIdx.x]] = index;
                }
            }

            if (index == numParticles - 1) {
                gridData.cellStop[hash] = index + 1;
            }

            uint sortedIndex = gridData.index[index];

            sortedData.position[index] = unsortedData.position[sortedIndex];
            sortedData.velocity[index] = unsortedData.velocity[sortedIndex];
            sortedData.veleval[index] = unsortedData.veleval[sortedIndex];

        }

        ////////////////////////////////////////////////////////////////////////

        template<class D>
        __global__ void computeDensity(
            uint numParticles,
            D sortedData,
            GridData gridData
        ) {
            uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
            if (index >= numParticles) {
                return;
            }

            float3 position = make_float3(sortedData.position[index]);

            Density::Data data;
            data.sorted = sortedData;

            iterateNeighbourCells<Neighbours<Density, Density::Data>, Density::Data>(
              index, position, data, gridData
            );
        }

        ////////////////////////////////////////////////////////////////////////

        template<class D>
        __global__ void computeForce(
            uint numParticles,
            D sortedData,
            GridData gridData
        ) {
            uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
            if (index >= numParticles) {
                return;
            }

            float3 position = make_float3(sortedData.position[index]);

            Force::Data data;
            data.sorted = sortedData;

            iterateNeighbourCells<Neighbours<Force, Force::Data>, Force::Data>(
                index, position, data, gridData
            );
        }

        ////////////////////////////////////////////////////////////////////////

    };
};

#endif // __SPH_KERNEL_CU__
