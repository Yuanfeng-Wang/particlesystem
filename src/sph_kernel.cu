#include "hip/hip_runtime.h"
#ifndef __SPH_KERNEL_CU__
#define __SPH_KERNEL_CU__

__constant__ SPH::FluidParams    cudaFluidParams;
__constant__ SPH::PrecalcParams  cudaPrecalcParams;

#include "boundary_walls.cu"
#include "grid.cuh"
#include "grid_utils.cu"
#include "sph_density.cu"
#include "sph_force.cu"
#include "sph_neighbours.cu"

namespace SPH {

    namespace Kernel {

        using namespace Grid::Utils;

        ////////////////////////////////////////////////////////////////////////

        template<class D>
        __global__ void integrate(
            int numParticles,
            float deltaTime,
            D data,
            D sortedData,
            GridData gridData
        ) {
            int index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
            if (index >= numParticles) {
                return;
            }

            float3 position = make_float3(sortedData.position[index]);
            float3 velocity = make_float3(sortedData.velocity[index]);
            float3 veleval = make_float3(sortedData.veleval[index]);

            float3 force = make_float3(sortedData.force[index]);
            //float pressure = sortedData.pressure[index];

            float3 externalForce = make_float3(0.0f, 0.0f, 0.0f);
            //externalForce.y -= 9.8f;

            // add no-penetration force due to "walls"
            externalForce += Boundary::Walls::calculateWallsNoPenetrationForce(
                    position, veleval,
                    cudaGridParams.min,
                    cudaGridParams.max,
                    cudaFluidParams.boundaryDistance,
                    cudaFluidParams.boundaryStiffness,
                    cudaFluidParams.boundaryDampening);

            // add no-slip force due to "walls"
            /*externalForce += Boundary::Walls::calculateWallsNoSlipForce(
                    position, veleval, force + externalForce,
                    cudaGridParams.min,
                    cudaGridParams.max,
                    cudaFluidParams.boundaryDistance,
                    cudaFluidParams.frictionKinetic/deltaTime,
                    cudaFluidParams.frictionStaticLimit);
            */
            float3 f = force + externalForce;

            float speed = length(force);

            if (speed > cudaFluidParams.velocityLimit) {
                f *= cudaFluidParams.velocityLimit / speed;
            }

            float3 vnext = velocity + f * deltaTime;
            veleval = vnext;//(velocity + vnext) * 0.5f;
            velocity = veleval;

            position += vnext * deltaTime;

            uint sortedIndex = gridData.index[index];

            /*if ((position.y - EPSILON) <= cudaGridParams.min.y) {
                position.y =  cudaGridParams.min.y + EPSILON;
            }

            if ((position.y + EPSILON) >= cudaGridParams.max.y) {
                position.y =  cudaGridParams.max.y - EPSILON;
            }*/


            data.position[sortedIndex] = make_float4(position, 1.0f);
            data.velocity[sortedIndex] = make_float4(velocity, 1.0f);
            data.veleval[sortedIndex] = make_float4(veleval, 1.0f);

            if (cudaFluidParams.dynamicColoring) {
                float3 color = (position - cudaGridParams.min) / cudaGridParams.size;
                data.color[sortedIndex] = make_float4(color, 1.0f);
            }

        }

        ////////////////////////////////////////////////////////////////////////


        // TODO this is the same for classical simulator, so place somewhere
        // where general codes are
        template<class D>
        __global__ void update (
            uint numParticles,
            D unsortedData,
            D sortedData,
            GridData gridData
        ) {
            uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
            if (index >= numParticles) {
                return;
            }

            extern __shared__ uint sharedHash[]; // blockSize + 1 elements

            uint hash = gridData.hash[index];

            sharedHash[threadIdx.x+1] = hash;

            if (index > 0 && threadIdx.x  == 0) {
                sharedHash[0] = gridData.hash[index-1];
            }

            __syncthreads();

            if (index == 0 || hash != sharedHash[threadIdx.x]) {
                gridData.cellStart[hash] = index;

                if (index > 0) {
                    gridData.cellStop[sharedHash[threadIdx.x]] = index;
                }
            }

            if (index == numParticles - 1) {
                gridData.cellStop[hash] = index + 1;
            }

            uint sortedIndex = gridData.index[index];

            sortedData.position[index] = unsortedData.position[sortedIndex];
            sortedData.velocity[index] = unsortedData.velocity[sortedIndex];

        }

        ////////////////////////////////////////////////////////////////////////

        template<class D>
        __global__ void computeDensity(
            uint numParticles,
            D sortedData,
            GridData gridData
        ) {
            uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
            if (index >= numParticles) {
                return;
            }

            float3 position = make_float3(sortedData.position[index]);

            Density::Data data;
            data.sorted = sortedData;

            iterateNeighbourCells<Neighbours<Density, Density::Data>, Density::Data>(
              index, position, data, gridData
            );
        }

        ////////////////////////////////////////////////////////////////////////

        template<class D>
        __global__ void computeForce(
            uint numParticles,
            D sortedData,
            GridData gridData
        ) {
            uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
            if (index >= numParticles) {
                return;
            }

            float3 position = make_float3(sortedData.position[index]);

            Force::Data data;
            data.sorted = sortedData;

            iterateNeighbourCells<Neighbours<Force, Force::Data>, Force::Data>(
                index, position, data, gridData
            );
        }

        ////////////////////////////////////////////////////////////////////////

    };
};

#endif // __SPH_KERNEL_CU__
