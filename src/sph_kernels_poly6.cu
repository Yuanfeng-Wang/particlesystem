#include "hip/hip_runtime.h"
#ifndef _SPH_KERNELS_POLY6_CU_
#define _SPH_KERNELS_POLY6_CU_

class Poly6 {

    public:

        ////////////////////////////////////////////////////////////////////////

        static __device__ __host__
        float getConstant (float smoothLen) {
            return 315.0f / (64.0f * M_PI * pow(smoothLen, 9.0f));
        }

        ////////////////////////////////////////////////////////////////////////

        static __device__ __host__
        float getVariable(float smoothLenSq, float3 r, float rLenSq) {
            float variableSquare = smoothLenSq - rLenSq;
            return variableSquare * variableSquare * variableSquare ;
        }

        ////////////////////////////////////////////////////////////////////////

        static __device__ __host__
        float getGradientConstant (float smoothLen) {
            return -945.0f / (32.0f * M_PI * pow(smoothLen, 9.0f));
        }

        ////////////////////////////////////////////////////////////////////////

        static __device__ __host__
        float3 getGradientVariable(float smoothLenSq, float3 r, float rLenSq) {
            float variableSquare = smoothLenSq - rLenSq;
            return r * (variableSquare * variableSquare);
        }

        ////////////////////////////////////////////////////////////////////////

        static __device__ __host__
        float3 getGradient(
            float smoothLen,
            float smoothLenSq,
            float3 r,
            float rLenSq
        ) {
            return
                getGradientConstant(smoothLen) *
                getGradientVariable(smoothLenSq, r, rLenSq);
        }

        ////////////////////////////////////////////////////////////////////////

        static __device__ __host__
        float getLaplacianConstant (float smoothLen) {
            return 945.0f / (8.0f * M_PI * pow(smoothLen, 9.0f));
        }

        ////////////////////////////////////////////////////////////////////////

        static __device__ __host__
        float getLaplacianVariable(float smoothLenSq, float rLenSq) {
            float variableSquare = smoothLenSq - rLenSq;
            return
            variableSquare * (rLenSq - ((3.0/4.0) * variableSquare));
        }

        ////////////////////////////////////////////////////////////////////////

};

#endif // _SPH_KERNELS_POLY6_CU_