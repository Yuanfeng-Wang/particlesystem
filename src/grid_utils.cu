#include "hip/hip_runtime.h"
#ifndef __GRID_UTILS_CU__
#define __GRID_UTILS_CU__

#include <cutil_math.h>

#include "grid.cuh"

namespace Grid {

    namespace Utils {

        ////////////////////////////////////////////////////////////////////////

        /**
         * Compute grid cell position
         *
         * @param position particle position
         * @param data grid parameters
         */
        inline __device__ int3 computeCellPosition(
            float3 const &position,
            GridParams const &params
        ) {
            // subtract grid_min (cell position) and multiply by delta
            return make_int3((position - params.min) * params.delta);
        }

        ////////////////////////////////////////////////////////////////////////

        /**
         * Compute grid cell hash
         *
         * @param position cell position
         * @param data grid parameters
         */
        inline __device__ uint computeCellHash(
            int3 const &position,
            GridParams const &params
        ) {

            int rx = (int) floor(params.resolution.x);
            int ry = (int) floor(params.resolution.y);
            int rz = (int) floor(params.resolution.z);

            // wrap grid... but since we can not assume size is power of 2
            // we can't use binary AND/& :/
            int px = position.x % rx;
            int py = position.y % ry;
            int pz = position.z % rz;

            if(px < 0) px += rx;
            if(py < 0) py += ry;
            if(pz < 0) pz += rz;

            // hash = x + y*width + z*width+height

            return
                px +
                __mul24(py, params.resolution.x) +
                __mul24(
                    params.resolution.x,
                    __umul24(pz, params.resolution.y)
                );

        }

        ////////////////////////////////////////////////////////////////////////

        template<class C, class D>
        __device__ void iterateCell(
            D &data,
            int3 const &cell,
            uint const &index,
            float3 const &position,
            GridData const &gridData
        ) {
            volatile uint hash = computeCellHash(cell, cudaGridParams);
            volatile uint cellStart = gridData.cellStart[hash];

            if (cellStart != EMPTY_CELL_VALUE) {
                volatile uint cellStop = gridData.cellStop[hash];

                for (uint indexN = cellStart; indexN<cellStop; indexN++) {
                    C::processNeighbour(data, index, indexN, position);
                }
            }
        }

        ////////////////////////////////////////////////////////////////////////

        template<class C, class D>
        __device__ void iterateNeighbourCells(
            uint const &index,
            float3 const &position,
            D &data,
            GridData const &gridData
        ) {

            C::preProcess(data, index);

            volatile int3 cell = computeCellPosition(position, cudaGridParams);

            for (uint z=cell.z-1; z<cell.z+1; z++) {
                for (uint y=cell.y-1; y<cell.y+1; y++) {
                    for (uint x=cell.x-1; x<cell.x+1; x++) {
                        iterateCell<C,D>(
                            data,
                            make_int3(x,y,z),
                            index,
                            position,
                            gridData
                        );
                    }
                }
            }

            C::postProcess(data, index);
        }
    };
};

#endif // __GRID_UTILS_CU__