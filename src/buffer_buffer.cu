#include "buffer_buffer.cuh"

#include <iostream>

namespace Buffer {

    ///////////////////////////////////////////////////////////////////////////

    template<class T> Buffer<T>::Buffer(Allocator* allocator, memory_t memory) {
        this->_allocator = allocator;
        this->_memory = memory;
        this->_bound = false;
        this->_size = 0;
        this->_memoryPtr = NULL;
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> Buffer<T>::~Buffer() {
        this->free();
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> error_t Buffer<T>::bind() {
        hipError_t hipError_t = hipBindTexture(
            0,
            this->_textureRef,
            this->_memoryPtr,
            this->_size*sizeof(T)
        );

        error_t error = parseCudaError(hipError_t);

        if (error == success) {
            this->_bound = true;
        }

        return error;
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> void Buffer<T>::unbind() {
        if (this->_bound) {
            hipUnbindTexture(this->_textureRef);
            this->_bound = false;
        }
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> error_t Buffer<T>::memset(int value) {
        size_t size = this->getMemorySize();
        error_t error = success;
        hipError_t hipError_t;

        if (size > 0) {
            switch (this->_memory) {
                case host:
                case hostPinned:
                    ::memset(this->_memoryPtr, value, size);
                    error = success;
                    break;
                case device:
                    hipError_t = hipMemset(this->_memoryPtr, value, size);
                    error = parseCudaError(hipError_t);
                    break;
                default:
                    error = unknownMemoryTypeError;
            }
        }

        return error;
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> void Buffer<T>::allocate(size_t size) {

        if (this->_size > 0 ) {
            if (size == this->_size) {
                return;
            } else {
                this->free();
            }
        }

        // calculate size of required memory
        size_t allocationSize = size * sizeof(T);

        error_t error =
            this->_allocator->allocate(
                (void **) &this->_memoryPtr,
                allocationSize,
                this->_memory
            );


        // TODO handle error
        if (error == success) {
            this->_size = size;
        }
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> void Buffer<T>::free() {
        if (this->_size > 0 ) {
            error_t error =
                this->_allocator->free(
                    (void**) &this->_memoryPtr,
                    this->_memory
                );

            // TODO handle error
            if (error == success) {
                this->_size = 0;
                this->_memoryPtr = NULL;
            }
        }
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> T* Buffer<T>::get() {
        return this->_memoryPtr;
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> size_t Buffer<T>::getSize() {
        return this->_size;
    }

    ///////////////////////////////////////////////////////////////////////////

    template<class T> size_t Buffer<T>::getMemorySize() {
        return this->_size * sizeof(T);
    }

    ///////////////////////////////////////////////////////////////////////////
}