#ifndef __BUFFER_MEMORY_CU__
#define __BUFFER_MEMORY_CU__

#include <iostream>

namespace Buffer {

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    Memory<T>::Memory(Allocator* allocator, memory_t memory) {
        this->_init(allocator, memory);
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    Memory<T>::Memory() {
        this->_init(new Allocator(), Device);
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    Memory<T>::~Memory() {
        this->free();
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    error_t Memory<T>::bind() {

        error_t error = Success;

        // TODO texture memory and fetch in kernels
        /*if (this->_memory == Device) {
            hipError_t hipError_t = hipBindTexture(
                0,
                this->_textureRef,
                this->_memoryPtr,
                this->_size*sizeof(T)
            );

            error = parseCudaError(hipError_t);
        }

        if (error == Success) {
            this->_bound = true;
        }*/

        return error;
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    void Memory<T>::unbind() {
        /*if (this->_bound) {
            if (this->_memory == Device) {
                hipUnbindTexture(this->_textureRef);
            }
            this->_bound = false;
        }*/
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    error_t Memory<T>::memset(int value) {
        size_t size = this->getMemorySize();
        error_t error = Success;
        hipError_t hipError_t;

        if (size > 0) {
            switch (this->_memory) {
                case Host:
                case HostPinned:
                    ::memset(this->_memoryPtr, value, size);
                    error = Success;
                    break;
                case Device:
                    hipError_t = hipMemset(this->_memoryPtr, value, size);
                    error = parseCudaError(hipError_t);
                    break;
                default:
                    error = UnknownMemoryTypeError;
            }
        }

        return error;
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    void Memory<T>::allocate(size_t size) {

        if (this->_size > 0 ) {
            if (size == this->_size) {
                return;
            } else {
                this->free();
            }
        }

        // calculate size of required memory
        size_t allocationSize = size * sizeof(T);

        error_t error =
            this->_allocator->allocate(
                (void **) &this->_memoryPtr,
                allocationSize,
                this->_memory
            );


        // TODO handle error
        if (error == Success) {
            this->_size = size;
        }
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    void Memory<T>::copyTo(void* dst, memory_t dstMem) {
        this->_copy(this->get(), dst, this->_memory, dstMem);
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    void Memory<T>::copyFrom(void* src, memory_t srcMem) {
        this->_copy(src, this->get(), srcMem, this->_memory);
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    void Memory<T>::free() {
        if (this->_size > 0 ) {
            error_t error =
                this->_allocator->free(
                    (void**) &this->_memoryPtr,
                    this->_memory
                );

            // TODO handle error
            if (error == Success) {
                this->_size = 0;
                this->_memoryPtr = NULL;
            }
        }
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    void Memory<T>::_init(Allocator* allocator, memory_t memory) {
        this->_allocator = allocator;
        this->_memory = memory;
        this->_bound = false;
        this->_size = 0;
        this->_memoryPtr = NULL;
    }

    ////////////////////////////////////////////////////////////////////////////

    template<class T>
    void Memory<T>::_copy(
        void* src,
        void* dst,
        memory_t srcMem,
        memory_t dstMem
    ) {

        enum hipMemcpyKind kind;

        if (srcMem == HostPinned) {
            srcMem = Host;
        }

        if (dstMem == HostPinned) {
            dstMem = Host;
        }

        if (srcMem == Device && dstMem == Device) {
            kind = hipMemcpyDeviceToDevice;
        } else if (srcMem == Device && dstMem == Host) {
            kind = hipMemcpyDeviceToHost;
        } else if (srcMem == Host && dstMem == Device) {
            kind = hipMemcpyHostToDevice;
        } else if (srcMem == Host && dstMem == Host) {
            kind = hipMemcpyHostToHost;
        } else {
            return;
        }

        cutilSafeCall(
            hipMemcpy(dst, src, this->_size * sizeof(T), kind)
        );
    }

    ////////////////////////////////////////////////////////////////////////////

}

#endif // __BUFFER_MEMORY_BUFFER_ĆU__